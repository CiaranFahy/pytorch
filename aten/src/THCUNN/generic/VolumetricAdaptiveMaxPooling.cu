
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/VolumetricAdaptiveMaxPooling.cu"
#else

#include <THCUNN/common.h>

// 5d tensor B x D x T x H x W

void THNN_(VolumetricAdaptiveMaxPooling_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           THCIndexTensor *indices,
           int osizeT,
           int osizeW,
           int osizeH)
{
  THCUNN_assertSameGPU(state, 3, input, output, indices);

  THCUNN_argCheck(state, !input->is_empty() && (input->dim() == 4 || input->dim() == 5), 2, input,
                  "4D or 5D (batch mode) tensor expected for input, but got: %s");

  THCIndex_t *indices_data;
  scalar_t *output_data;
  scalar_t *input_data;

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t istrideD, istrideT, istrideH, istrideW;
  int64_t totalZ;

  if (input->dim() == 4) {
    sizeD = input->size(0);
    isizeT = input->size(1);
    isizeH = input->size(2);
    isizeW = input->size(3);

    istrideD = input->stride(0);
    istrideT = input->stride(1);
    istrideH = input->stride(2);
    istrideW = input->stride(3);

    THCTensor_(resize4d)(state, output, sizeD, osizeT, osizeH, osizeW);
    THCIndexTensor_(resize4d)(state, indices, sizeD, osizeT, osizeH, osizeW);

    totalZ = sizeD * osizeT;
  } else {
    input = THCTensor_(newContiguous)(state, input);

    int64_t sizeB = input->size(0);
    sizeD = input->size(1);
    isizeT = input->size(2);
    isizeH = input->size(3);
    isizeW = input->size(4);

    istrideD = input->stride(1);
    istrideT = input->stride(2);
    istrideH = input->stride(3);
    istrideW = input->stride(4);

    THCTensor_(resize5d)(state, output, sizeB, sizeD, osizeT, osizeH, osizeW);
    THCIndexTensor_(resize5d)(state, indices, sizeB, sizeD, osizeT, osizeH, osizeW);

    totalZ = sizeB * sizeD * osizeT;
  }

  input_data = THCTensor_(data)(state, input);
  output_data = THCTensor_(data)(state, output);
  indices_data = THCIndexTensor_(data)(state, indices);

  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);
    cunn_VolumetricAdaptiveMaxPooling_updateOutput_kernel
      <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
        input_data, output_data, indices_data, isizeT, isizeH, isizeW,
        osizeT, osizeH, osizeW, istrideD, istrideT, istrideH, istrideW, offsetZ
      );

    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }

  if (input->dim() == 5) {
    // clean
    THCTensor_(free)(state, input);
  }
}

void THNN_(VolumetricAdaptiveMaxPooling_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCIndexTensor *indices)
{
  THCUNN_assertSameGPU(state, 4, input, indices, gradOutput, gradInput);

  gradOutput = THCTensor_(newContiguous)(state, gradOutput);

  THCTensor_(resizeAs)(state, gradInput, input);
  THCTensor_(zero)(state, gradInput);

  THCIndex_t *indices_data;
  scalar_t *gradInput_data;
  scalar_t *gradOutput_data;

  int64_t sizeD, isizeT, isizeH, isizeW;
  int64_t osizeT, osizeH, osizeW;
  int64_t totalZ;

  if (input->dim() == 4) {
    sizeD = input->size(0);
    isizeT = input->size(1);
    isizeH = input->size(2);
    isizeW = input->size(3);

    osizeT = gradOutput->size(1);
    osizeH = gradOutput->size(2);
    osizeW = gradOutput->size(3);
  } else {
    sizeD = input->size(1);
    isizeT = input->size(2);
    isizeH = input->size(3);
    isizeW = input->size(4);

    osizeT = gradOutput->size(2);
    osizeH = gradOutput->size(3);
    osizeW = gradOutput->size(4);
  }

  bool atomic = (isizeW%osizeW != 0) || (isizeH%osizeH != 0) || (isizeT%osizeT != 0);

  if (input->dim() == 4) {
    totalZ = sizeD * osizeT;
  } else {
    int sizeB = input->size(0);
    totalZ = sizeB * sizeD * osizeT;
  }

  indices_data = THCIndexTensor_(data)(state, indices);
  gradInput_data = THCTensor_(data)(state, gradInput);
  gradOutput_data = THCTensor_(data)(state, gradOutput);

  int64_t offsetZ = 0;
  dim3 threads(32, 8);
  // each H*W plane is processed by blocksH thread blocks
  int blocksH = max((int)(16L / totalZ), 1);
  while (totalZ > 0) {
    dim3 blocks(totalZ > 65535 ? 65535 : totalZ, blocksH);

    if (atomic)
    {
      cunn_atomic_VolumetricAdaptiveMaxPooling_updateGradInput_kernel
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data, gradOutput_data, indices_data,
          isizeT, isizeH, isizeW, osizeT, osizeH, osizeW, offsetZ
        );
    } else {
      cunn_VolumetricAdaptiveMaxPooling_updateGradInput_kernel
        <<<blocks, threads, 0, THCState_getCurrentStream(state)>>>(
          gradInput_data, gradOutput_data, indices_data,
          isizeT, isizeH, isizeW, osizeT, osizeH, osizeW, offsetZ
        );
    }

    totalZ -= 65535;
    offsetZ += 65535;
    THCudaCheck(cudaGetLastError());
  }
  // clean
  THCTensor_(free)(state, gradOutput);
}

#endif
